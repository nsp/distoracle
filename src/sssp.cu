#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <vector>

#include "types.h"
#include "qt.hpp"

using namespace std;

#define MAX_THREADS_PER_BLOCK 512

__global__ void
DijkstraKernel1(uint32  no_of_nodes,
                uint32  no_of_edges,
                uint32 *g_graph_nodes,
                uint32 *g_graph_edges,
                uint32 *g_graph_weights,
                uint32 *g_up_cost,
                char   *g_graph_mask,
                uint32 *g_cost) {
  int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
  int i,end,id;
  if(tid<no_of_nodes && g_graph_mask[tid]) {
    end = (tid < no_of_nodes-1) ? g_graph_nodes[tid+1] : no_of_edges;
    for(i = g_graph_nodes[tid]; i< end; i++) {
      id = g_graph_edges[i];
      atomicMin(&g_up_cost[id], g_cost[tid]+g_graph_weights[i]);
    }
    g_graph_mask[tid]=false;
  }
}

__global__ void
DijkstraKernel2(uint32  no_of_nodes,
                uint32 *g_up_cost,
                char   *g_graph_mask,
                uint32 *g_cost,
                char   *d_finished) {
  int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
  if(tid<no_of_nodes && g_cost[tid] > g_up_cost[tid]) {
    g_cost[tid] = g_up_cost[tid];
    g_graph_mask[tid] = true;
    *d_finished = true;
  }
  if(tid<no_of_nodes) g_up_cost[tid] = g_cost[tid];
}

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                                    \
    hipError_t _m_cudaStat = value;                                  \
    if (_m_cudaStat != hipSuccess) {                                 \
        fprintf(stderr, "Error %s at line %d in file %s\n",           \
                hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
        exit(1);                                                      \
    } }

struct device_ptrs {
  uint32 *up_cost;
  uint32 *cost;
  uint32 *nodes;
  uint32 *edges;
  uint32 *evals;
  char   *mask;
  char   *finished;
};

void graph_to_dev(device_ptrs dps,
                  uint32 nn, uint32 *nodes,
                  uint32 ne, uint32 *edges, uint32 *evals) {
  CUDA_CHECK_RETURN( hipMemcpy( dps.nodes, nodes, 4*nn, hipMemcpyHostToDevice) );
  CUDA_CHECK_RETURN( hipMemcpy( dps.edges, edges, 4*ne, hipMemcpyHostToDevice) );
  CUDA_CHECK_RETURN( hipMemcpy( dps.evals, evals, 4*ne, hipMemcpyHostToDevice) );
}

void prob_to_dev(device_ptrs dps,
                 uint32 nn, uint32 *up_cost, uint32 *cost, char *mask) {
  CUDA_CHECK_RETURN( hipMemcpy( dps.up_cost, up_cost, 4*nn, hipMemcpyHostToDevice) );
  CUDA_CHECK_RETURN( hipMemcpy( dps.cost,    cost,    4*nn, hipMemcpyHostToDevice) );
  CUDA_CHECK_RETURN( hipMemcpy( dps.mask,    mask,    nn, hipMemcpyHostToDevice) );
}

void sssp( device_ptrs dps,
           const uint32 no_of_nodes,
           const uint32 *h_graph_nodes,
                 uint32 *h_up_cost,
                 char   *h_mask,
           const uint32 no_of_edges,
           const uint32 *h_graph_edges,
           const uint32 *h_graph_weights,
           const uint32 source_id,
           uint32 *h_cost) {
  cout << "in sssp..."; cout.flush();
  const uint32 MAX_COST = 1 << 30;

  for( uint32 i=0; i<no_of_nodes; i++) {
    h_up_cost[i] = MAX_COST;
    h_cost[i]    = MAX_COST;
    h_mask[i]    = false;
  }

  h_cost[source_id] = 0;
  h_mask[source_id] = true;

  // Copy lists to device memory
  prob_to_dev(dps, no_of_nodes, h_up_cost, h_cost, h_mask);

  //make a char to check if the execution is over
  char finished;

  // setup execution parameters
  // Make execution Parameters according to the number of nodes
  // Distribute threads across multiple Blocks if necessary
  uint32 num_of_blocks = 1;
  uint32 num_of_threads_per_block = no_of_nodes;
  if(no_of_nodes>MAX_THREADS_PER_BLOCK) {
    num_of_blocks = (uint32)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK);
    num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
  }
  dim3  grid( num_of_blocks, 1, 1);
  dim3  threads( num_of_threads_per_block, 1, 1);

  cout << "running kernels..."; cout.flush();
  uint32 k=0;
  do {
    DijkstraKernel1<<< grid, threads, 0 >>>( no_of_nodes,
                                             no_of_edges,
                                             dps.nodes,
                                             dps.edges,
                                             dps.evals,
                                             dps.up_cost,
                                             dps.mask,
                                             dps.cost );
    k++;
    finished = false;
    CUDA_CHECK_RETURN( hipMemcpy( dps.finished, &finished, 1, hipMemcpyHostToDevice ) );
    DijkstraKernel2<<< grid, threads, 0 >>>( no_of_nodes,
                                             dps.up_cost,
                                             dps.mask,
                                             dps.cost,
                                             dps.finished);
    CUDA_CHECK_RETURN( hipDeviceSynchronize() );    // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN( hipGetLastError() );
    CUDA_CHECK_RETURN( hipMemcpy( &finished, dps.finished, 1, hipMemcpyDeviceToHost ) );
  } while( finished );
  cout << "done in " << k << " iterations..."; cout.flush();
  // copy result from device to host
  CUDA_CHECK_RETURN( hipMemcpy( h_cost, dps.cost, 4*no_of_nodes, hipMemcpyDeviceToHost) );

}

int32 main( int32 argc, char** argv) {

  uint32 nn = 0;
  uint32 ne = 0;

  printf("Reading File\n");
  // Read in Graph from a file
  FILE *fp = fopen("/home/natep/cuda-workspace/sssp/NY.out","r");
  if(!fp) {
    printf("Error Reading graph file\n");
    return -1;
  }

  int32 source_id = 0;

  fscanf(fp,"%d",&nn);
  printf("No of Nodes: %d\n",nn);

  // allocate host memory
  uint32 *h_graph_nodes, *h_up_cost;
  char   *h_mask;
  CUDA_CHECK_RETURN( hipHostMalloc( &h_graph_nodes, sizeof(uint32)*nn ) );
  CUDA_CHECK_RETURN( hipHostMalloc( &h_up_cost,     sizeof(uint32)*nn ) );
  CUDA_CHECK_RETURN( hipHostMalloc( &h_mask,        nn ) );

  // initalize the memory
  uint32 start, edgeno;
  for( uint32 i = 0; i < nn; i++ ) {
    fscanf(fp,"%d %d",&start,&edgeno);
    h_graph_nodes[i] = start;
  }

  //read the source int from the file
  fscanf(fp,"%d",&source_id);
  printf("Source vid: %d\n", source_id);

  fscanf(fp,"%d",&ne);
  printf("No of Edges: %d\n", ne);

  uint32 id;
  uint32* h_graph_edges, *h_graph_weights;
  CUDA_CHECK_RETURN( hipHostMalloc( &h_graph_edges, sizeof(uint32)*ne ) );
  CUDA_CHECK_RETURN( hipHostMalloc( &h_graph_weights, sizeof(uint32)*ne ) );
  for(uint32 i=0; i < ne ; i++) {
    fscanf(fp,"%d",&id);
    h_graph_edges[i] = id;
    fscanf(fp,"%d",&id);
    h_graph_weights[i] = id;
  }

  if(fp) fclose(fp);

  printf("Read File\n");
  printf("Avg Branching Factor: %f\n",ne/(float)nn);

  // allocate mem for the result on host side
  uint32 *h_cost;
  CUDA_CHECK_RETURN( hipHostMalloc( &h_cost, sizeof(uint32)*nn ) );

  // allocate everything in device
  device_ptrs dps;
  CUDA_CHECK_RETURN( hipMalloc( &dps.up_cost, 4*nn ) );
  CUDA_CHECK_RETURN( hipMalloc( &dps.cost,    4*nn ) );
  CUDA_CHECK_RETURN( hipMalloc( &dps.nodes,   4*nn ) );
  CUDA_CHECK_RETURN( hipMalloc( &dps.edges,   4*ne ) );
  CUDA_CHECK_RETURN( hipMalloc( &dps.evals,   4*ne ) );
  CUDA_CHECK_RETURN( hipMalloc( &dps.mask,    nn ) );
  CUDA_CHECK_RETURN( hipMalloc( &dps.finished,1));
  graph_to_dev( dps,
                nn, h_graph_nodes,
                ne, h_graph_edges, h_graph_weights);

  /********************************************************************************/

  ifstream cof("/home/natep/cuda-workspace/sssp/NY.co");
  string v;
  uint32 lat, lon;
  uint32 max = 1U << 28;
  vector<Qvtx*> qvtxes;
  qvtxes.reserve(nn);
  Qt qt;
  for(uint32 i=0; i<nn; i++) {
    cof >> v >> id >> lat >> lon;
    qvtxes.push_back(new Qvtx(i, morton_code(max+lat, max+lon)));
    qt.insert(qvtxes.at(i));
  }
  cof.close();

  printf("all read, first=%lu, qt.size=%lu\n", qvtxes[0]->z, qt.size());

  /********************************************************************************/

  //Store the result into a file
  ofstream rf("/home/natep/cuda-workspace/sssp/result.txt");
  double eps = 0.5;
  double sep = 2/eps;
  std::deque<std::pair<qblck, qblck> > Q;
  qblck root = QBLCK(0, 0);
  qt.childpairs(root, Q);
  uint64 qiters = 0;
  while(!Q.empty()) {
    qblck a = Q.front().first;
    qblck b = Q.front().second;
    cout << ++qiters << "/" << Q.size() << ": ";
    cout << "a=" << LEVEL_OF_QBLCK(a) << "|" << hex << CODE_OF_QBLCK(a) << dec << ", ";
    cout << "b=" << LEVEL_OF_QBLCK(b) << "|" << hex << CODE_OF_QBLCK(b) << dec << endl;
    cerr << qiters << endl;
    Q.pop_front();
    if(a==b && qt.isnotleaf(a)) {
      cout << " Same" << endl;
      qt.childpairs(a, Q);
    } else {
      // Choose rep point of A
      Qvtx *pa = qt.getRep(a);
      if(NULL == pa) {
        cout << "nonexistent node ended up in q as a" << endl;
        continue;
      }
      // Get sssp from pa
      cout << " sssp(" << pa->vid << "|" << pa->z <<")..."; cout.flush();;
      sssp(dps,
           nn, h_graph_nodes, h_up_cost, h_mask,
           ne, h_graph_edges, h_graph_weights,
           pa->vid,
           h_cost);
      cout << "done" << endl;
      // Measure diameter of A
      uint32 da = qt.netdiam(h_cost, a);
      // Choose rep point of B
      Qvtx *pb = qt.getRep(b);
      if(NULL == pa) {
        cout << "nonexistent node ended up in q as b" << endl;
        continue;
      }
      // dg = graph_dist(pa, pb)
      uint32 dg_a_b = h_cost[pb->vid];
      // Get sssp from pb
      cout << " sssp(" << pb->vid << "|" << pb->z <<")..."; cout.flush();;
      sssp(dps,
           nn, h_graph_nodes, h_up_cost, h_mask,
           ne, h_graph_edges, h_graph_weights,
           pb->vid,
           h_cost);
      cout << "done" << endl;
      // Measure diameter of B
      uint32 db = qt.netdiam(h_cost, b);
      // r = max(da, db)
      uint32 r = std::max(da, db);
      // if dg/r >= sep
      if( dg_a_b/(double)r >= sep ) {
        cout << " L: " << hex << CODE_OF_QBLCK(a) << " -> " << CODE_OF_QBLCK(b) << " = " << dec << dg_a_b << endl;
        rf << CODE_OF_QBLCK(a) << " -> " << CODE_OF_QBLCK(b) << " = " << dg_a_b << endl;
      } else {
        cout << " ~L" << endl;
        std::vector<qblck> la, lb;
        if(qt.isnotleaf(a)) {
          for(uint64 cn=0; cn<4; cn++) {
            if(qt.contains(cn)) {
              la.push_back(child(a, cn));
            } else {
              la.push_back(a);
            }
          }
        }
        if(qt.isnotleaf(b)) {
          for(uint64 cn=0; cn<4; cn++) {
            if(qt.contains(cn)) {
              la.push_back(child(b, cn));
            } else {
              la.push_back(b);
            }
          }
        }
        for(std::vector<qblck>::iterator ca = la.begin(); ca != la.end(); ca++) {
          for(std::vector<qblck>::iterator cb = lb.begin(); cb != lb.end(); cb++) {
            Q.push_back(std::make_pair(*ca, *cb));
          }
        }
      }
    }
  }
  
  // for(uint32 i=0; i<0; i++) {
  //   source_id = rand() % nn;
  //   printf("source = %d\n", source_id);
  //   sssp( dps,
  //         nn, h_graph_nodes, h_up_cost, h_mask,
  //         ne, h_graph_edges, h_graph_weights,
  //         source_id,
  //         h_cost );
  // }

  /********************************************************************************/

  printf("Computation finished\n");

  //Store the result into a file
  rf.close();
  printf("Result stored in result.txt\n");

  // cleanup memory  
  CUDA_CHECK_RETURN(hipHostFree(h_graph_nodes));
  CUDA_CHECK_RETURN(hipHostFree(h_graph_edges));
  CUDA_CHECK_RETURN(hipHostFree(h_graph_weights));
  CUDA_CHECK_RETURN(hipHostFree(h_mask));
  CUDA_CHECK_RETURN(hipHostFree(h_up_cost));
  CUDA_CHECK_RETURN(hipHostFree(h_cost));
  CUDA_CHECK_RETURN(hipFree(dps.nodes));
  CUDA_CHECK_RETURN(hipFree(dps.edges));
  CUDA_CHECK_RETURN(hipFree(dps.mask));
  CUDA_CHECK_RETURN(hipFree(dps.evals));
  CUDA_CHECK_RETURN(hipFree(dps.up_cost));
  CUDA_CHECK_RETURN(hipFree(dps.cost));
  CUDA_CHECK_RETURN(hipFree(dps.finished));
  CUDA_CHECK_RETURN(hipDeviceReset());
  return 0;
}
